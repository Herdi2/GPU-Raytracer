#include "hip/hip_runtime.h"
#include "cudart/hip/hip_vector_types.h"
#include "cudart/cuda_math.h"

#include "Common.h"

__device__ __constant__ int screen_width;
__device__ __constant__ int screen_pitch;
__device__ __constant__ int screen_height;

#include "Util.h"
#include "Shading.h"
#include "Sky.h"
#include "Random.h"

#define INFINITY ((float)(1e+300 * 1e+300))

// Frame Buffers
__device__ float4 * frame_buffer_albedo;
__device__ float4 * frame_buffer_direct;
__device__ float4 * frame_buffer_indirect;

__device__ float4 * frame_buffer_moment;

// GBuffers (OpenGL resource-mapped textures)
__device__ Texture<float4> gbuffer_normal_and_depth;
__device__ Texture<float2> gbuffer_uv;
__device__ Texture<float4> gbuffer_uv_gradient;
__device__ Texture<int2>   gbuffer_mesh_id_and_triangle_id;
__device__ Texture<float2> gbuffer_screen_position_prev;
__device__ Texture<float2> gbuffer_depth_gradient;

// SVGF History Buffers (Temporally Integrated)
__device__ int    * history_length;
__device__ float4 * history_direct;
__device__ float4 * history_indirect;
__device__ float4 * history_moment;
__device__ float4 * history_normal_and_depth;

// Used for Temporal Anti-Aliasing
__device__ float4 * taa_frame_curr;
__device__ float4 * taa_frame_prev;

// Final Frame buffer, shared with OpenGL
__device__ Surface<float4> accumulator; 

#include "SVGF.h"
#include "TAA.h"

// Vector3 buffer in SoA layout
struct Vector3_SoA {
	float * x;
	float * y;
	float * z;

	__device__ void from_float3(int index, const float3 & vector) {
		x[index] = vector.x;
		y[index] = vector.y;
		z[index] = vector.z;
	}

	__device__ float3 to_float3(int index) const {
		return make_float3(
			x[index],
			y[index],
			z[index]
		);
	}
};

struct HitBuffer {
	float4 * hits;

	__device__ void set(int index, int mesh_id, int triangle_id, float u, float v) {
		hits[index] = make_float4(uint_as_float(mesh_id), uint_as_float(triangle_id), u, v);
	}

	__device__ void get(int index, int & mesh_id, int & triangle_id, float & u, float & v) const {
		float4 hit = __ldg(&hits[index]);

		mesh_id     = float_as_uint(hit.x);
		triangle_id = float_as_uint(hit.y);
		u = hit.z;
		v = hit.w;
	}
};

// Input to the Trace and Sort Kernels in SoA layout
struct TraceBuffer {
	Vector3_SoA origin;
	Vector3_SoA direction;
	
	HitBuffer hits;

	int       * pixel_index;
	Vector3_SoA throughput;

	char  * last_material_type;
	float * last_pdf;
};

// Input to the various Shade Kernels in SoA layout
struct MaterialBuffer {
	Vector3_SoA direction;	
	
	HitBuffer hits;

	int       * pixel_index;
	Vector3_SoA throughput;
};

// Input to the Shadow Trace Kernel in SoA layout
struct ShadowRayBuffer {
	Vector3_SoA ray_origin;
	Vector3_SoA ray_direction;

	float * max_distance;

	int       * pixel_index;
	Vector3_SoA illumination;
};

__device__ TraceBuffer     ray_buffer_trace;
__device__ MaterialBuffer  ray_buffer_shade_diffuse;
__device__ MaterialBuffer  ray_buffer_shade_dielectric;
__device__ MaterialBuffer  ray_buffer_shade_glossy;
__device__ ShadowRayBuffer ray_buffer_shadow;

// Number of elements in each Buffer
// Sizes are stored for ALL bounces so we only have to reset these
// values back to 0 after every frame, instead of after every bounce
struct BufferSizes {
	int trace     [NUM_BOUNCES];
	int diffuse   [NUM_BOUNCES];
	int dielectric[NUM_BOUNCES];
	int glossy    [NUM_BOUNCES];
	int shadow    [NUM_BOUNCES];

	// Global counters for tracing kernels
	int rays_retired       [NUM_BOUNCES];
	int rays_retired_shadow[NUM_BOUNCES];
} __device__ buffer_sizes;

#include "Tracing.h"

// Sends the rasterized GBuffer to the right Material kernels,
// as if the primary Rays they were Raytraced 
extern "C" __global__ void kernel_primary(
	int rand_seed,
	int sample_index,
	int pixel_offset,
	int pixel_count,
	bool jitter,
	float3 camera_position,
	float3 camera_bottom_left_corner,
	float3 camera_x_axis,
	float3 camera_y_axis
) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= pixel_count) return;

	int index_offset = index + pixel_offset;
	int x = index_offset % screen_width;
	int y = index_offset / screen_width;

	int pixel_index = x + y * screen_pitch;

	unsigned seed = (pixel_index + rand_seed * 199494991) * 949525949;

	float u_screenspace = float(x) + 0.5f;
	float v_screenspace = float(y) + 0.5f;

	float u = u_screenspace / float(screen_width);
	float v = v_screenspace / float(screen_height);

	float2 uv          = gbuffer_uv         .get(u, v);
	float4 uv_gradient = gbuffer_uv_gradient.get(u, v);

	int2 mesh_id_and_triangle_id = gbuffer_mesh_id_and_triangle_id.get(u, v);
	int mesh_id     = mesh_id_and_triangle_id.x;
	int triangle_id = mesh_id_and_triangle_id.y - 1;

	float dx = 0.0f;
	float dy = 0.0f;
	
	if (jitter) {
		// Jitter the barycentric coordinates in screen space using their screen space differentials
		dx = random_float_heitz(x, y, sample_index, 0, 0, seed) - 0.5f;
		dy = random_float_heitz(x, y, sample_index, 0, 1, seed) - 0.5f;

		uv.x = saturate(uv.x + uv_gradient.x * dx + uv_gradient.z * dy);
		uv.y = saturate(uv.y + uv_gradient.y * dx + uv_gradient.w * dy);
	}

	float3 ray_direction = normalize(camera_bottom_left_corner
		+ (u_screenspace + dx) * camera_x_axis
		+ (v_screenspace + dy) * camera_y_axis
	);

	// Triangle ID -1 means no hit
	if (triangle_id == -1) {
		frame_buffer_albedo[pixel_index] = make_float4(1.0f);
		frame_buffer_direct[pixel_index] = make_float4(sample_sky(ray_direction));

		return;
	}

	const Material & material = materials[triangle_get_material_id(triangle_id)];

	// Decide which Kernel to invoke, based on Material Type
	switch (material.type) {
		case Material::Type::LIGHT: {
			// Terminate Path
			frame_buffer_albedo[pixel_index] = make_float4(1.0f);
			frame_buffer_direct[pixel_index] = make_float4(material.emission);

			break;
		}
		
		case Material::Type::DIFFUSE: {
			int index_out = atomic_agg_inc(&buffer_sizes.diffuse[0]);

			ray_buffer_shade_diffuse.direction.from_float3(index_out, ray_direction);

			ray_buffer_shade_diffuse.hits.set(index_out, mesh_id, triangle_id, uv.x, uv.y);

			ray_buffer_shade_diffuse.pixel_index[index_out] = pixel_index;
			ray_buffer_shade_diffuse.throughput.from_float3(index_out, make_float3(1.0f));

			break;
		}

		case Material::Type::DIELECTRIC: {
			int index_out = atomic_agg_inc(&buffer_sizes.dielectric[0]);

			ray_buffer_shade_dielectric.direction.from_float3(index_out, ray_direction);

			ray_buffer_shade_dielectric.hits.set(index_out, mesh_id, triangle_id, uv.x, uv.y);

			ray_buffer_shade_dielectric.pixel_index[index_out] = pixel_index;
			ray_buffer_shade_dielectric.throughput.from_float3(index_out, make_float3(1.0f));
			
			break;
		}

		case Material::Type::GLOSSY: {
			int index_out = atomic_agg_inc(&buffer_sizes.glossy[0]);

			ray_buffer_shade_glossy.direction.from_float3(index_out, ray_direction);

			ray_buffer_shade_glossy.hits.set(index_out, mesh_id, triangle_id, uv.x, uv.y);

			ray_buffer_shade_glossy.pixel_index[index_out] = pixel_index;
			ray_buffer_shade_glossy.throughput.from_float3(index_out, make_float3(1.0f));
			
			break;
		}
	}
}

extern "C" __global__ void kernel_generate(
	int rand_seed,
	int sample_index,
	int pixel_offset,
	int pixel_count,
	float3 camera_position,
	float3 camera_bottom_left_corner,
	float3 camera_x_axis,
	float3 camera_y_axis
) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= pixel_count) return;

	int index_offset = index + pixel_offset;
	int x = index_offset % screen_width;
	int y = index_offset / screen_width;

	unsigned seed = (index_offset + rand_seed * 199494991) * 949525949;

	int pixel_index = x + y * screen_pitch;
	ASSERT(pixel_index < screen_pitch * screen_height, "Pixel should fit inside the buffer");

	// Add random value between 0 and 1 so that after averaging we get anti-aliasing
	float u = float(x) + random_float_heitz(x, y, sample_index, 0, 0, seed);
	float v = float(y) + random_float_heitz(x, y, sample_index, 0, 1, seed);

	// Create primary Ray that starts at the Camera's position and goes through the current pixel
	ray_buffer_trace.origin   .from_float3(index, camera_position);
	ray_buffer_trace.direction.from_float3(index, normalize(camera_bottom_left_corner
		+ u * camera_x_axis
		+ v * camera_y_axis
	));

	ray_buffer_trace.pixel_index[index] = pixel_index;
	ray_buffer_trace.throughput.from_float3(index, make_float3(1.0f));

	ray_buffer_trace.last_material_type[index] = char(Material::Type::DIELECTRIC);
}

extern "C" __global__ void kernel_trace(int bounce) {
	bvh_trace(buffer_sizes.trace[bounce], &buffer_sizes.rays_retired[bounce]);
}

extern "C" __global__ void kernel_sort(int rand_seed, int bounce) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= buffer_sizes.trace[bounce]) return;

	float3 ray_origin    = ray_buffer_trace.origin   .to_float3(index);
	float3 ray_direction = ray_buffer_trace.direction.to_float3(index);

	int   hit_mesh_id;
	int   hit_triangle_id;
	float hit_u;
	float hit_v;
	ray_buffer_trace.hits.get(index, hit_mesh_id, hit_triangle_id, hit_u, hit_v);

	int    ray_pixel_index = ray_buffer_trace.pixel_index[index];
	float3 ray_throughput  = ray_buffer_trace.throughput.to_float3(index);

	// If we didn't hit anything, sample the Sky
	if (hit_triangle_id == -1) {
		float3 illumination = ray_throughput * sample_sky(ray_direction);

		if (bounce == 0) {
			frame_buffer_albedo[ray_pixel_index] = make_float4(1.0f);
			frame_buffer_direct[ray_pixel_index] = make_float4(illumination);
		} else if (bounce == 1) {
			frame_buffer_direct[ray_pixel_index] += make_float4(illumination);
		} else {
			frame_buffer_indirect[ray_pixel_index] += make_float4(illumination);
		}

		return;
	}

	// Get the Material of the Triangle we hit
	const Material & material = materials[triangle_get_material_id(hit_triangle_id)];

	if (material.type == Material::Type::LIGHT) {
#if ENABLE_NEXT_EVENT_ESTIMATION
		bool no_mis = 
			(ray_buffer_trace.last_material_type[index] == char(Material::Type::DIELECTRIC)) ||
			(ray_buffer_trace.last_material_type[index] == char(Material::Type::GLOSSY) && material.roughness < ROUGHNESS_CUTOFF);
#else
		bool no_mis = true;
#endif
		if (no_mis) {
			float3 illumination = ray_throughput * material.emission;

			if (bounce == 0) {
				frame_buffer_albedo[ray_pixel_index] = make_float4(1.0f);
				frame_buffer_direct[ray_pixel_index] = make_float4(material.emission);
			} else if (bounce == 1) {
				frame_buffer_direct[ray_pixel_index] += make_float4(illumination);
			} else {
				frame_buffer_indirect[ray_pixel_index] += make_float4(illumination);
			}

			return;
		}

#if ENABLE_MULTIPLE_IMPORTANCE_SAMPLING
		float3 light_position_0, light_position_edge_1, light_position_edge_2;
		float3 light_normal_0,   light_normal_edge_1,   light_normal_edge_2;

		triangle_get_positions_and_normals(hit_triangle_id,
			light_position_0, light_position_edge_1, light_position_edge_2,
			light_normal_0,   light_normal_edge_1,   light_normal_edge_2
		);

		float3 light_point  = barycentric(hit_u, hit_v, light_position_0, light_position_edge_1, light_position_edge_2);
		float3 light_normal = barycentric(hit_u, hit_v, light_normal_0,   light_normal_edge_1,   light_normal_edge_2);
	
		light_normal = normalize(light_normal);
	
		float3 to_light = light_point - ray_origin;;
		float distance_to_light_squared = dot(to_light, to_light);
		float distance_to_light         = sqrtf(distance_to_light_squared);
	
		// ray_direction is the same direction as light is same direction as to_light, but normalized
		to_light = ray_direction;

		float cos_o = fabsf(dot(to_light, light_normal));

		// if (cos_o <= 0.0f) return;

		float light_area = 0.5f * length(cross(light_position_edge_1, light_position_edge_2));
		
		float brdf_pdf = ray_buffer_trace.last_pdf[index];

#if LIGHT_SELECTION == LIGHT_SELECT_UNIFORM
		float light_select_pdf = 1.0f / float(light_count);
#elif LIGHT_SELECTION == LIGHT_SELECT_AREA
		float light_select_pdf = light_area / light_area_total;
#endif
		float light_pdf = light_select_pdf * distance_to_light_squared / (cos_o * light_area); // 1 / solid angle

		float mis_pdf = brdf_pdf + light_pdf;

		float3 illumination = ray_throughput * material.emission * brdf_pdf / mis_pdf;

		if (bounce == 1) {
			frame_buffer_direct[ray_pixel_index] += make_float4(illumination);
		} else {
			frame_buffer_indirect[ray_pixel_index] += make_float4(illumination);
		}
#endif

		return;
	}

	unsigned seed = (index + rand_seed * 906313609) * 341828143;

	// Russian Roulette
	float p_survive = saturate(fmaxf(ray_throughput.x, fmaxf(ray_throughput.y, ray_throughput.z)));
	if (random_float_xorshift(seed) > p_survive) {
		return;
	}

	ray_throughput /= p_survive;

	switch (material.type) {
		case Material::Type::DIFFUSE: {
			int index_out = atomic_agg_inc(&buffer_sizes.diffuse[bounce]);

			ray_buffer_shade_diffuse.direction.from_float3(index_out, ray_direction);

			ray_buffer_shade_diffuse.hits.set(index_out, hit_mesh_id, hit_triangle_id, hit_u, hit_v);

			ray_buffer_shade_diffuse.pixel_index[index_out] = ray_buffer_trace.pixel_index[index];
			ray_buffer_shade_diffuse.throughput.from_float3(index_out, ray_throughput);

			break;
		}

		case Material::Type::DIELECTRIC: {
			int index_out = atomic_agg_inc(&buffer_sizes.dielectric[bounce]);

			ray_buffer_shade_dielectric.direction.from_float3(index_out, ray_direction);

			ray_buffer_shade_dielectric.hits.set(index_out, hit_mesh_id, hit_triangle_id, hit_u, hit_v);

			ray_buffer_shade_dielectric.pixel_index[index_out] = ray_buffer_trace.pixel_index[index];
			ray_buffer_shade_dielectric.throughput.from_float3(index_out, ray_throughput);

			break;
		}

		case Material::Type::GLOSSY: {
			int index_out = atomic_agg_inc(&buffer_sizes.glossy[bounce]);

			ray_buffer_shade_glossy.direction.from_float3(index_out, ray_direction);

			ray_buffer_shade_glossy.hits.set(index_out, hit_mesh_id, hit_triangle_id, hit_u, hit_v);

			ray_buffer_shade_glossy.pixel_index[index_out] = ray_buffer_trace.pixel_index[index];
			ray_buffer_shade_glossy.throughput.from_float3(index_out, ray_throughput);

			break;
		}
	}
}

extern "C" __global__ void kernel_shade_diffuse(int rand_seed, int bounce, int sample_index) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= buffer_sizes.diffuse[bounce]) return;

	float3 ray_direction = ray_buffer_shade_diffuse.direction.to_float3(index);

	int   ray_mesh_id;
	int   ray_triangle_id;
	float ray_u;
	float ray_v;
	ray_buffer_shade_diffuse.hits.get(index, ray_mesh_id, ray_triangle_id, ray_u, ray_v);

	int ray_pixel_index = ray_buffer_shade_diffuse.pixel_index[index];
	int x = ray_pixel_index % screen_pitch;
	int y = ray_pixel_index / screen_pitch;

	float3 ray_throughput = ray_buffer_shade_diffuse.throughput.to_float3(index);

	ASSERT(ray_triangle_id != -1, "Ray must have hit something for this Kernel to be invoked!");

	unsigned seed = (index + rand_seed * 794454497) * 781939187;

	const Material & material = materials[triangle_get_material_id(ray_triangle_id)];

	ASSERT(material.type == Material::Type::DIFFUSE, "Material should be diffuse in this Kernel");

	float3 hit_triangle_position_0,  hit_triangle_position_edge_1,  hit_triangle_position_edge_2;
	float3 hit_triangle_normal_0,    hit_triangle_normal_edge_1,    hit_triangle_normal_edge_2;
	float2 hit_triangle_tex_coord_0, hit_triangle_tex_coord_edge_1, hit_triangle_tex_coord_edge_2;

	triangle_get_positions_normals_and_tex_coords(ray_triangle_id,
		hit_triangle_position_0,  hit_triangle_position_edge_1,  hit_triangle_position_edge_2,
		hit_triangle_normal_0,    hit_triangle_normal_edge_1,    hit_triangle_normal_edge_2,
		hit_triangle_tex_coord_0, hit_triangle_tex_coord_edge_1, hit_triangle_tex_coord_edge_2
	);

	float3 hit_point_model  = barycentric(ray_u, ray_v, hit_triangle_position_0,  hit_triangle_position_edge_1,  hit_triangle_position_edge_2);
	float3 hit_normal_model = barycentric(ray_u, ray_v, hit_triangle_normal_0,    hit_triangle_normal_edge_1,    hit_triangle_normal_edge_2);
	float2 hit_tex_coord    = barycentric(ray_u, ray_v, hit_triangle_tex_coord_0, hit_triangle_tex_coord_edge_1, hit_triangle_tex_coord_edge_2);

	float3 hit_point;  // World space
	float3 hit_normal; // World space
	mesh_transform_point_and_normal(ray_mesh_id, hit_point_model, hit_normal_model, hit_point, hit_normal);
	
	hit_normal = normalize(hit_normal);
	if (dot(ray_direction, hit_normal) > 0.0f) hit_normal = -hit_normal;

	float3 albedo     = material.albedo(hit_tex_coord.x, hit_tex_coord.y);
	float3 throughput = ray_throughput * albedo;

	if (bounce == 0) {
		frame_buffer_albedo[ray_pixel_index] = make_float4(albedo);
	}

#if ENABLE_NEXT_EVENT_ESTIMATION
	if (light_count > 0) {
		// Trace Shadow Ray
		float light_u, light_v;
		int   light_id = random_point_on_random_light(x, y, sample_index, bounce, seed, light_u, light_v);

		float3 light_position_0, light_position_edge_1, light_position_edge_2;
		float3 light_normal_0,   light_normal_edge_1,   light_normal_edge_2;

		triangle_get_positions_and_normals(light_id,
			light_position_0, light_position_edge_1, light_position_edge_2,
			light_normal_0,   light_normal_edge_1,   light_normal_edge_2
		);

		float3 light_point  = barycentric(light_u, light_v, light_position_0, light_position_edge_1, light_position_edge_2);
		float3 light_normal = barycentric(light_u, light_v, light_normal_0,   light_normal_edge_1,   light_normal_edge_2);
	
		float3 to_light = light_point - hit_point;
		float distance_to_light_squared = dot(to_light, to_light);
		float distance_to_light         = sqrtf(distance_to_light_squared);

		// Normalize the vector to the light
		to_light /= distance_to_light;

		float cos_o = -dot(to_light, light_normal);
		float cos_i =  dot(to_light,   hit_normal);
	
		// Only trace Shadow Ray if light transport is possible given the normals
		if (cos_o > 0.0f && cos_i > 0.0f) {
			// NOTE: N dot L is included here
			float brdf     = cos_i * ONE_OVER_PI;
			float brdf_pdf = cos_i * ONE_OVER_PI;

			float light_area = 0.5f * length(cross(light_position_edge_1, light_position_edge_2));

#if LIGHT_SELECTION == LIGHT_SELECT_UNIFORM
			float light_select_pdf = 1.0f / float(light_count);
#elif LIGHT_SELECTION == LIGHT_SELECT_AREA
			float light_select_pdf = light_area / light_area_total;
#endif
			float light_pdf = light_select_pdf * distance_to_light_squared / (cos_o * light_area); // 1 / solid angle

#if ENABLE_MULTIPLE_IMPORTANCE_SAMPLING
			float mis_pdf = brdf_pdf + light_pdf;
#else
			float mis_pdf = light_pdf;
#endif

			float3 emission     = materials[triangle_get_material_id(light_id)].emission;
			float3 illumination = throughput * brdf * emission / mis_pdf;

			int shadow_ray_index = atomic_agg_inc(&buffer_sizes.shadow[bounce]);

			ray_buffer_shadow.ray_origin   .from_float3(shadow_ray_index, hit_point);
			ray_buffer_shadow.ray_direction.from_float3(shadow_ray_index, to_light);

			ray_buffer_shadow.max_distance[shadow_ray_index] = distance_to_light - EPSILON;

			ray_buffer_shadow.pixel_index[shadow_ray_index] = ray_pixel_index;
			ray_buffer_shadow.illumination.from_float3(shadow_ray_index, illumination);
		}
	}
#endif

	if (bounce == NUM_BOUNCES - 1) return;

	int index_out = atomic_agg_inc(&buffer_sizes.trace[bounce + 1]);

	float3 direction = random_cosine_weighted_direction(x, y, sample_index, bounce, seed, hit_normal);

	ray_buffer_trace.origin   .from_float3(index_out, hit_point);
	ray_buffer_trace.direction.from_float3(index_out, direction);

	ray_buffer_trace.pixel_index[index_out]  = ray_pixel_index;
	ray_buffer_trace.throughput.from_float3(index_out, throughput);

	ray_buffer_trace.last_material_type[index_out] = char(Material::Type::DIFFUSE);
	ray_buffer_trace.last_pdf[index_out] = fabsf(dot(direction, hit_normal)) * ONE_OVER_PI;
}

extern "C" __global__ void kernel_shade_dielectric(int rand_seed, int bounce) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= buffer_sizes.dielectric[bounce] || bounce == NUM_BOUNCES - 1) return;

	float3 ray_direction = ray_buffer_shade_dielectric.direction.to_float3(index);

	int   ray_mesh_id;
	int   ray_triangle_id;
	float ray_u;
	float ray_v;
	ray_buffer_shade_dielectric.hits.get(index, ray_mesh_id, ray_triangle_id, ray_u, ray_v);

	int ray_pixel_index = ray_buffer_shade_dielectric.pixel_index[index];

	float3 ray_throughput = ray_buffer_shade_dielectric.throughput.to_float3(index);

	ASSERT(ray_triangle_id != -1, "Ray must have hit something for this Kernel to be invoked!");

	unsigned seed = (index + rand_seed * 758505857) * 364686463;

	const Material & material = materials[triangle_get_material_id(ray_triangle_id)];

	ASSERT(material.type == Material::Type::DIELECTRIC, "Material should be dielectric in this Kernel");

	float3 hit_triangle_position_0, hit_triangle_position_edge_1, hit_triangle_position_edge_2;
	float3 hit_triangle_normal_0,   hit_triangle_normal_edge_1,   hit_triangle_normal_edge_2;

	triangle_get_positions_and_normals(ray_triangle_id,
		hit_triangle_position_0, hit_triangle_position_edge_1, hit_triangle_position_edge_2,
		hit_triangle_normal_0,   hit_triangle_normal_edge_1,   hit_triangle_normal_edge_2
	);

	float3 hit_point_model  = barycentric(ray_u, ray_v, hit_triangle_position_0, hit_triangle_position_edge_1, hit_triangle_position_edge_2);
	float3 hit_normal_model = barycentric(ray_u, ray_v, hit_triangle_normal_0,   hit_triangle_normal_edge_1,   hit_triangle_normal_edge_2);
	
	hit_normal_model = normalize(hit_normal_model);

	float3 hit_point;  // World space
	float3 hit_normal; // World space
	mesh_transform_point_and_normal(ray_mesh_id, hit_point_model, hit_normal_model, hit_point, hit_normal);

	int index_out = atomic_agg_inc(&buffer_sizes.trace[bounce + 1]);

	float3 direction;
	float3 direction_reflected = reflect(ray_direction, hit_normal);

	float3 normal;
	float  cos_theta;

	float n_1;
	float n_2;

	float dir_dot_normal = dot(ray_direction, hit_normal);
	if (dir_dot_normal < 0.0f) { 
		// Entering material		
		n_1 = 1.0f;
		n_2 = material.index_of_refraction;

		normal    =  hit_normal;
		cos_theta = -dir_dot_normal;
	} else { 
		// Leaving material
		n_1 = material.index_of_refraction;
		n_2 = 1.0f;

		normal    = -hit_normal;
		cos_theta =  dir_dot_normal;
	}

	float eta = n_1 / n_2;
	float k = 1.0f - eta*eta * (1.0f - cos_theta*cos_theta);

	if (k < 0.0f) {
		// Total Internal Reflection
		direction = direction_reflected;
	} else {
		float3 direction_refracted = normalize(eta * ray_direction + (eta * cos_theta - sqrtf(k)) * hit_normal);

		float fresnel = fresnel_schlick(n_1, n_2, cos_theta, -dot(direction_refracted, normal));

		if (random_float_xorshift(seed) < fresnel) {
			direction = direction_reflected;
		} else {
			direction = direction_refracted;
		}
	}

	if (bounce == 0) {
		frame_buffer_albedo[ray_pixel_index] = make_float4(1.0f);
	}

	ray_buffer_trace.origin   .from_float3(index_out, hit_point);
	ray_buffer_trace.direction.from_float3(index_out, direction);

	ray_buffer_trace.pixel_index[index_out] = ray_pixel_index;
	ray_buffer_trace.throughput.from_float3(index_out, ray_throughput);

	ray_buffer_trace.last_material_type[index_out] = char(Material::Type::DIELECTRIC);
}

extern "C" __global__ void kernel_shade_glossy(int rand_seed, int bounce, int sample_index) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= buffer_sizes.glossy[bounce]) return;

	float3 direction_in = -1.0f * ray_buffer_shade_glossy.direction.to_float3(index);

	int   ray_mesh_id;
	int   ray_triangle_id;
	float ray_u;
	float ray_v;
	ray_buffer_shade_glossy.hits.get(index, ray_mesh_id, ray_triangle_id, ray_u, ray_v);

	int ray_pixel_index = ray_buffer_shade_glossy.pixel_index[index];
	int x = ray_pixel_index % screen_pitch;
	int y = ray_pixel_index / screen_pitch; 

	float3 ray_throughput = ray_buffer_shade_glossy.throughput.to_float3(index);

	ASSERT(ray_triangle_id != -1, "Ray must have hit something for this Kernel to be invoked!");

	unsigned seed = (index + rand_seed * 354767453) * 346434643;

	const Material & material = materials[triangle_get_material_id(ray_triangle_id)];

	ASSERT(material.type == Material::Type::GLOSSY, "Material should be glossy in this Kernel");

	float3 hit_triangle_position_0,  hit_triangle_position_edge_1,  hit_triangle_position_edge_2;
	float3 hit_triangle_normal_0,    hit_triangle_normal_edge_1,    hit_triangle_normal_edge_2;
	float2 hit_triangle_tex_coord_0, hit_triangle_tex_coord_edge_1, hit_triangle_tex_coord_edge_2;

	triangle_get_positions_normals_and_tex_coords(ray_triangle_id,
		hit_triangle_position_0,  hit_triangle_position_edge_1,  hit_triangle_position_edge_2,
		hit_triangle_normal_0,    hit_triangle_normal_edge_1,    hit_triangle_normal_edge_2,
		hit_triangle_tex_coord_0, hit_triangle_tex_coord_edge_1, hit_triangle_tex_coord_edge_2
	);

	float3 hit_point_model  = barycentric(ray_u, ray_v, hit_triangle_position_0,  hit_triangle_position_edge_1,  hit_triangle_position_edge_2);
	float3 hit_normal_model = barycentric(ray_u, ray_v, hit_triangle_normal_0,    hit_triangle_normal_edge_1,    hit_triangle_normal_edge_2);
	float2 hit_tex_coord    = barycentric(ray_u, ray_v, hit_triangle_tex_coord_0, hit_triangle_tex_coord_edge_1, hit_triangle_tex_coord_edge_2);

	float3 hit_point;  // World space
	float3 hit_normal; // World space
	mesh_transform_point_and_normal(ray_mesh_id, hit_point_model, hit_normal_model, hit_point, hit_normal);

	float3 albedo = material.albedo(hit_tex_coord.x, hit_tex_coord.y);
	float3 throughput = ray_throughput * albedo;

	if (bounce == 0) {
		frame_buffer_albedo[ray_pixel_index] = make_float4(albedo);
	}

	// Slightly widen the distribution to prevent the weights from becoming too large (see Walter et al. 2007)
	float alpha = (1.2f - 0.2f * sqrtf(dot(direction_in, hit_normal))) * material.roughness;
	
#if ENABLE_NEXT_EVENT_ESTIMATION
	if (light_count > 0 && material.roughness >= ROUGHNESS_CUTOFF) {
		// Trace Shadow Ray
		float light_u;
		float light_v;
		int light_id = random_point_on_random_light(x, y, sample_index, bounce, seed, light_u, light_v);

		float3 light_position_0, light_position_edge_1, light_position_edge_2;
		float3 light_normal_0,   light_normal_edge_1,   light_normal_edge_2;

		triangle_get_positions_and_normals(light_id,
			light_position_0, light_position_edge_1, light_position_edge_2,
			light_normal_0,   light_normal_edge_1,   light_normal_edge_2
		);

		float3 light_point  = barycentric(light_u, light_v, light_position_0, light_position_edge_1, light_position_edge_2);
		float3 light_normal = barycentric(light_u, light_v, light_normal_0,   light_normal_edge_1,   light_normal_edge_2);

		float3 to_light = light_point - hit_point;
		float distance_to_light_squared = dot(to_light, to_light);
		float distance_to_light         = sqrtf(distance_to_light_squared);

		// Normalize the vector to the light
		to_light /= distance_to_light;

		float cos_o = -dot(to_light, light_normal);
		float cos_i =  dot(to_light,   hit_normal);

		// Only trace Shadow Ray if light transport is possible given the normals
		if (cos_o > 0.0f && cos_i > 0.0f) {
			float3 half_vector = normalize(to_light + direction_in);

			float i_dot_n = dot(direction_in, hit_normal);
			float m_dot_n = dot(half_vector,  hit_normal);

			float F = fresnel_schlick(material.index_of_refraction, 1.0f, i_dot_n, i_dot_n);
			float D = microfacet_D(m_dot_n, alpha);
			float G = microfacet_G(i_dot_n, cos_i, i_dot_n, cos_i, m_dot_n, alpha);

			// NOTE: N dot L is omitted from the denominator here
			float brdf     = (F * G * D) / (4.0f * i_dot_n);
			float brdf_pdf = F * D * m_dot_n / (4.0f * dot(half_vector, direction_in));
			
			float light_area = 0.5f * length(cross(light_position_edge_1, light_position_edge_2));

#if LIGHT_SELECTION == LIGHT_SELECT_UNIFORM
			float light_select_pdf = 1.0f / float(light_count);
#elif LIGHT_SELECTION == LIGHT_SELECT_AREA
			float light_select_pdf = light_area / light_area_total;
#endif
			float light_pdf = light_select_pdf * distance_to_light_squared / (cos_o * light_area); // 1 / solid angle

#if ENABLE_MULTIPLE_IMPORTANCE_SAMPLING
			float mis_pdf = brdf_pdf + light_pdf;
#else
			float mis_pdf = light_pdf;
#endif
			float3 emission     = materials[triangle_get_material_id(light_id)].emission;
			float3 illumination = throughput * brdf * emission / mis_pdf;

			int shadow_ray_index = atomic_agg_inc(&buffer_sizes.shadow[bounce]);

			ray_buffer_shadow.ray_origin   .from_float3(shadow_ray_index, hit_point);
			ray_buffer_shadow.ray_direction.from_float3(shadow_ray_index, to_light);

			ray_buffer_shadow.max_distance[shadow_ray_index] = distance_to_light - EPSILON;

			ray_buffer_shadow.pixel_index[shadow_ray_index] = ray_pixel_index;
			ray_buffer_shadow.illumination.from_float3(shadow_ray_index, illumination);
		}
	}
#endif

	if (bounce == NUM_BOUNCES - 1) return;

	hit_normal = normalize(hit_normal);
	if (dot(direction_in, hit_normal) < 0.0f) hit_normal = -hit_normal;

	// Sample normal distribution in spherical coordinates
	float theta = atanf(sqrtf(-alpha * alpha * logf(random_float_heitz(x, y, sample_index, bounce, 4, seed) + 1e-8f)));
	float phi   = TWO_PI * random_float_heitz(x, y, sample_index, bounce, 5, seed);

	float sin_theta, cos_theta;
	float sin_phi,   cos_phi;

	sincos(theta, &sin_theta, &cos_theta);
	sincos(phi,   &sin_phi,   &cos_phi);

	// Convert from spherical coordinates to cartesian coordinates
	float3 micro_normal_local = make_float3(sin_theta * cos_phi, sin_theta * sin_phi, cos_theta);

	float3 hit_tangent, hit_binormal;
	orthonormal_basis(hit_normal, hit_tangent, hit_binormal);

	float3 micro_normal_world = local_to_world(micro_normal_local, hit_tangent, hit_binormal, hit_normal);

	float3 direction_out = reflect(-direction_in, micro_normal_world);

	float i_dot_m = dot(direction_in,  micro_normal_world);
	float o_dot_m = dot(direction_out, micro_normal_world);
	float i_dot_n = dot(direction_in,       hit_normal);
	float o_dot_n = dot(direction_out,      hit_normal);
	float m_dot_n = dot(micro_normal_world, hit_normal);

	float F = fresnel_schlick(material.index_of_refraction, 1.0f, i_dot_m, i_dot_m);
	float D = microfacet_D(m_dot_n, alpha);
	float G = microfacet_G(i_dot_m, o_dot_m, i_dot_n, o_dot_n, m_dot_n, alpha);
	float weight = fabsf(i_dot_m) * F * G / fabsf(i_dot_n * m_dot_n);

	int index_out = atomic_agg_inc(&buffer_sizes.trace[bounce + 1]);

	ray_buffer_trace.origin   .from_float3(index_out, hit_point);
	ray_buffer_trace.direction.from_float3(index_out, direction_out);

	ray_buffer_trace.pixel_index[index_out] = ray_pixel_index;
	ray_buffer_trace.throughput.from_float3(index_out, throughput);

	ray_buffer_trace.last_material_type[index_out] = char(Material::Type::GLOSSY);
	ray_buffer_trace.last_pdf[index_out] = D * fabsf(m_dot_n) / (4.0f * fabsf(o_dot_m));
}

extern "C" __global__ void kernel_shadow_trace(int bounce) {
	bvh_trace_shadow(buffer_sizes.shadow[bounce], &buffer_sizes.rays_retired_shadow[bounce], bounce);
}

extern "C" __global__ void kernel_accumulate(bool demodulate_albedo, float frames_since_camera_moved) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= screen_width || y >= screen_height) return;

	int pixel_index = x + y * screen_pitch;

	float4 direct   = frame_buffer_direct  [pixel_index];
	float4 indirect = frame_buffer_indirect[pixel_index];

	float4 colour = direct + indirect;

	if (demodulate_albedo) {
		colour /= fmaxf(frame_buffer_albedo[pixel_index], make_float4(1e-8f));
	}

	if (frames_since_camera_moved > 0.0f) {
		float4 colour_prev = accumulator.get(x, y);

		// Take average over n samples by weighing the current content of the framebuffer by (n-1) and the new sample by 1
		colour = (colour_prev * (frames_since_camera_moved - 1.0f) + colour) / frames_since_camera_moved;
	}

	accumulator.set(x, y, colour);

	// @SPEED
	// Clear frame buffers for next frame
	frame_buffer_albedo  [pixel_index] = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
	frame_buffer_direct  [pixel_index] = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
	frame_buffer_indirect[pixel_index] = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
}
