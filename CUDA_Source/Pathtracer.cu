#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>
#include <corecrt_math.h>

#include "cuda_math.h"

#include "../Common.h"

#include "Tracing.h"
#include "Lighting.h"
#include "Sky.h"
#include "Util.h"

surface<void, 2> frame_buffer;
surface<void, 2> accumulator;

__device__ void frame_buffer_add(int x, int y, const float3 & colour) {
	float4 prev;
	surf2Dread<float4>(&prev, frame_buffer, x * sizeof(float4), y);
	
	surf2Dwrite<float4>(prev + make_float4(colour, 0.0f), frame_buffer, x * sizeof(float4), y, hipBoundaryModeClamp);
}

struct RayBuffer {
	float3 * origin;
	float3 * direction;
	
	int   * triangle_id;
	float * u;
	float * v;

	int    * pixel_index;
	float3 * throughput;

	char * last_material_type;
};

__device__ RayBuffer ray_buffer_extend;
__device__ RayBuffer ray_buffer_shade_diffuse;
__device__ RayBuffer ray_buffer_shade_dielectric;
__device__ RayBuffer ray_buffer_shade_glossy;

struct ShadowRayBuffer {
	int   * triangle_id;
	float * u;
	float * v;

	int    * pixel_index;
	float3 * throughput;
};

__device__ ShadowRayBuffer shadow_ray_buffer;

__device__ int N_ext;
__device__ int N_diffuse;
__device__ int N_dielectric;
__device__ int N_glossy;
__device__ int N_shadow;

extern "C" __global__ void kernel_generate(
	int rand_seed,
	float3 camera_position,
	float3 camera_top_left_corner,
	float3 camera_x_axis,
	float3 camera_y_axis
) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= SCREEN_WIDTH * SCREEN_HEIGHT) return;

	unsigned seed = (index + rand_seed * 199494991) * 949525949;
	
	const int BLOCK_WIDTH  = 8;
	const int BLOCK_HEIGHT = 4;
	const int BLOCK_SIZE   = BLOCK_WIDTH * BLOCK_HEIGHT;

	int block_index = index / BLOCK_SIZE;
	int i = (block_index % (SCREEN_WIDTH / BLOCK_WIDTH)) * BLOCK_WIDTH;
	int j = (block_index / (SCREEN_WIDTH / BLOCK_WIDTH)) * BLOCK_HEIGHT;

	ASSERT(i < SCREEN_WIDTH, "");
	ASSERT(j < SCREEN_HEIGHT, "");

	int k = (index % BLOCK_SIZE) % BLOCK_WIDTH;
	int l = (index % BLOCK_SIZE) / BLOCK_WIDTH;

	ASSERT(k < BLOCK_WIDTH, "");
	ASSERT(l < BLOCK_HEIGHT, "");

	int x = i + k;
	int y = j + l;

	ASSERT(x < SCREEN_WIDTH, "");
	ASSERT(y < SCREEN_HEIGHT, "");

	int pixel_index = x + y * SCREEN_WIDTH;

	// Add random value between 0 and 1 so that after averaging we get anti-aliasing
	float u = x + random_float(seed);
	float v = y + random_float(seed);

	ASSERT(pixel_index < SCREEN_WIDTH * SCREEN_HEIGHT, "Pixel should be on screen");

	// Create primary Ray that starts at the Camera's position and goes trough the current pixel
	ray_buffer_extend.origin[index]    = camera_position;
	ray_buffer_extend.direction[index] = normalize(camera_top_left_corner
		+ u * camera_x_axis
		+ v * camera_y_axis
	);
	
	ray_buffer_extend.pixel_index[index] = pixel_index;
	ray_buffer_extend.throughput[index]  = make_float3(1.0f);

	ray_buffer_extend.last_material_type[index] = char(Material::Type::DIELECTRIC);
}

extern "C" __global__ void kernel_extend() {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= N_ext) return;

	float3 ray_origin    = ray_buffer_extend.origin[index];
	float3 ray_direction = ray_buffer_extend.direction[index];
	
	Ray ray;
	ray.origin    = ray_origin;
	ray.direction = ray_direction;
	ray.direction_inv = make_float3(
		1.0f / ray.direction.x, 
		1.0f / ray.direction.y, 
		1.0f / ray.direction.z
	);

	RayHit hit;
	bvh_trace(ray, hit);

	if (hit.t == INFINITY) {
		int ray_pixel_index = ray_buffer_extend.pixel_index[index];

		int x = ray_pixel_index % SCREEN_WIDTH;
		int y = ray_pixel_index / SCREEN_WIDTH; 

		frame_buffer_add(x, y, ray_buffer_extend.throughput[index] * sample_sky(ray_direction));

		return;
	}

	Material::Type material_type = materials[triangles_material_id[hit.triangle_id]].type;
	
	if (material_type == Material::Type::DIFFUSE) {
		int index_out = atomic_agg_inc(&N_diffuse);

		// ray_buffer_shade_diffuse.direction[index_out] = ray_direction;

		ray_buffer_shade_diffuse.triangle_id[index_out] = hit.triangle_id;
		ray_buffer_shade_diffuse.u[index_out] = hit.u;
		ray_buffer_shade_diffuse.v[index_out] = hit.v;

		ray_buffer_shade_diffuse.pixel_index[index_out] = ray_buffer_extend.pixel_index[index];
		ray_buffer_shade_diffuse.throughput[index_out]  = ray_buffer_extend.throughput[index];

		ray_buffer_shade_diffuse.last_material_type[index_out] = ray_buffer_extend.last_material_type[index];
	} else if (material_type == Material::Type::DIELECTRIC) {
		int index_out = atomic_agg_inc(&N_dielectric);

		ray_buffer_shade_dielectric.direction[index_out] = ray_direction;

		ray_buffer_shade_dielectric.triangle_id[index_out] = hit.triangle_id;
		ray_buffer_shade_dielectric.u[index_out] = hit.u;
		ray_buffer_shade_dielectric.v[index_out] = hit.v;

		ray_buffer_shade_dielectric.pixel_index[index_out] = ray_buffer_extend.pixel_index[index];
		ray_buffer_shade_dielectric.throughput[index_out]  = ray_buffer_extend.throughput[index];

		ray_buffer_shade_dielectric.last_material_type[index_out] = ray_buffer_extend.last_material_type[index];
	} else if (material_type == Material::Type::GLOSSY) {
		int index_out = atomic_agg_inc(&N_glossy);

		ray_buffer_shade_glossy.direction[index_out] = ray_direction;

		ray_buffer_shade_glossy.triangle_id[index_out] = hit.triangle_id;
		ray_buffer_shade_glossy.u[index_out] = hit.u;
		ray_buffer_shade_glossy.v[index_out] = hit.v;

		ray_buffer_shade_glossy.pixel_index[index_out] = ray_buffer_extend.pixel_index[index];
		ray_buffer_shade_glossy.throughput[index_out]  = ray_buffer_extend.throughput[index];

		ray_buffer_shade_glossy.last_material_type[index_out] = ray_buffer_extend.last_material_type[index];
	}
}

extern "C" __global__ void kernel_shade_diffuse(int rand_seed, int bounce) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= N_diffuse) return;

	// float3 ray_direction = ray_buffer_shade_diffuse.direction[index];

	int   ray_triangle_id = ray_buffer_shade_diffuse.triangle_id[index];
	float ray_u = ray_buffer_shade_diffuse.u[index];
	float ray_v = ray_buffer_shade_diffuse.v[index];

	int    ray_pixel_index = ray_buffer_shade_diffuse.pixel_index[index];
	float3 ray_throughput  = ray_buffer_shade_diffuse.throughput[index];

	Material::Type ray_last_material_type = Material::Type(ray_buffer_shade_diffuse.last_material_type[index]);

	int x = ray_pixel_index % SCREEN_WIDTH;
	int y = ray_pixel_index / SCREEN_WIDTH; 

	ASSERT(ray_triangle_id != -1, "Ray must have hit something for this Kernel to be invoked!");

	unsigned seed = (ray_pixel_index + rand_seed * 312080213) * 781939187;

	const Material & material = materials[triangles_material_id[ray_triangle_id]];

	if (light_count > 0) {
		if (material.is_light()) {
			if (ray_last_material_type == Material::Type::DIELECTRIC) {
				frame_buffer_add(x, y, ray_throughput * material.emittance);
			}

			return;
		}

		int shadow_ray_index = atomic_agg_inc(&N_shadow);

		shadow_ray_buffer.triangle_id[shadow_ray_index] = ray_triangle_id;
		shadow_ray_buffer.u[shadow_ray_index] = ray_u;
		shadow_ray_buffer.v[shadow_ray_index] = ray_v;

		shadow_ray_buffer.pixel_index[shadow_ray_index] = ray_pixel_index;
		shadow_ray_buffer.throughput[shadow_ray_index]  = ray_throughput;
	}

	// Russian Roulette termination
	if (bounce > 3) {
		float one_minus_p = fmaxf(ray_throughput.x, fmaxf(ray_throughput.y, ray_throughput.z));
		if (random_float(seed) > one_minus_p) {
			return;
		}

		ray_throughput /= one_minus_p;
	}

	float3 hit_point     = barycentric(ray_u, ray_v, triangles_position0 [ray_triangle_id], triangles_position_edge1 [ray_triangle_id], triangles_position_edge2 [ray_triangle_id]);
	float3 hit_normal    = barycentric(ray_u, ray_v, triangles_normal0   [ray_triangle_id], triangles_normal_edge1   [ray_triangle_id], triangles_normal_edge2   [ray_triangle_id]);
	float2 hit_tex_coord = barycentric(ray_u, ray_v, triangles_tex_coord0[ray_triangle_id], triangles_tex_coord_edge1[ray_triangle_id], triangles_tex_coord_edge2[ray_triangle_id]);

	// if (dot(ray_direction, hit_normal) > 0.0f) hit_normal = -hit_normal;

	int index_out = atomic_agg_inc(&N_ext);

	ray_buffer_extend.origin[index_out]    = hit_point;
	ray_buffer_extend.direction[index_out] = cosine_weighted_diffuse_reflection(seed, hit_normal);

	ray_buffer_extend.pixel_index[index_out] = ray_pixel_index;
	ray_buffer_extend.throughput[index_out]  = ray_throughput * material.albedo(hit_tex_coord.x, hit_tex_coord.y);

	ray_buffer_extend.last_material_type[index_out] = char(Material::Type::DIFFUSE);
}

extern "C" __global__ void kernel_shade_dielectric(int rand_seed, int bounce) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= N_dielectric) return;

	float3 ray_direction = ray_buffer_shade_dielectric.direction[index];

	int   ray_triangle_id = ray_buffer_shade_dielectric.triangle_id[index];
	float ray_u = ray_buffer_shade_dielectric.u[index];
	float ray_v = ray_buffer_shade_dielectric.v[index];

	int    ray_pixel_index = ray_buffer_shade_dielectric.pixel_index[index];
	float3 ray_throughput  = ray_buffer_shade_dielectric.throughput[index];

	Material::Type ray_last_material_type = Material::Type(ray_buffer_shade_dielectric.last_material_type[index]);

	int x = ray_pixel_index % SCREEN_WIDTH;
	int y = ray_pixel_index / SCREEN_WIDTH; 

	ASSERT(ray_triangle_id != -1, "Ray must have hit something for this Kernel to be invoked!");

	unsigned seed = (ray_pixel_index + rand_seed * 312080213) * 781939187;

	const Material & material = materials[triangles_material_id[ray_triangle_id]];

	if (light_count > 0) {
		if (material.is_light()) {
			if (ray_last_material_type == Material::Type::DIELECTRIC) {
				frame_buffer_add(x, y, ray_throughput * material.emittance);
			}

			return;
		}
	}

	// Russian Roulette termination
	if (bounce > 3) {
		float one_minus_p = fmaxf(ray_throughput.x, fmaxf(ray_throughput.y, ray_throughput.z));
		if (random_float(seed) > one_minus_p) {
			return;
		}

		ray_throughput /= one_minus_p;
	}

	float3 hit_point     = barycentric(ray_u, ray_v, triangles_position0 [ray_triangle_id], triangles_position_edge1 [ray_triangle_id], triangles_position_edge2 [ray_triangle_id]);
	float3 hit_normal    = barycentric(ray_u, ray_v, triangles_normal0   [ray_triangle_id], triangles_normal_edge1   [ray_triangle_id], triangles_normal_edge2   [ray_triangle_id]);
	float2 hit_tex_coord = barycentric(ray_u, ray_v, triangles_tex_coord0[ray_triangle_id], triangles_tex_coord_edge1[ray_triangle_id], triangles_tex_coord_edge2[ray_triangle_id]);

	// if (dot(ray_direction, hit_normal) > 0.0f) hit_normal = -hit_normal;

	int index_out = atomic_agg_inc(&N_ext);

	float3 direction;
	float3 direction_reflected = reflect(ray_direction, hit_normal);

	float3 normal;
	float  cos_theta;

	float n_1;
	float n_2;

	float dir_dot_normal = dot(ray_direction, hit_normal);
	if (dir_dot_normal < 0.0f) { 
		// Entering material		
		n_1 = 1.0f;
		n_2 = material.index_of_refraction;

		normal    =  hit_normal;
		cos_theta = -dir_dot_normal;
	} else { 
		// Leaving material
		n_1 = material.index_of_refraction;
		n_2 = 1.0f;

		normal    = -hit_normal;
		cos_theta =  dir_dot_normal;
	}

	float eta = n_1 / n_2;
	float k = 1.0f - eta*eta * (1.0f - cos_theta*cos_theta);

	if (k < 0.0f) {
		direction = direction_reflected;
	} else {
		float3 direction_refracted = normalize(eta * ray_direction + (eta * cos_theta - sqrtf(k)) * hit_normal);

		// Use Schlick's Approximation
		float r_0 = (n_1 - n_2) / (n_1 + n_2);
		r_0 *= r_0;

		if (n_1 > n_2) {
			cos_theta = -dot(direction_refracted, normal);
		}

		float one_minus_cos         = 1.0f - cos_theta;
		float one_minus_cos_squared = one_minus_cos * one_minus_cos;

		float F_r = r_0 + ((1.0f - r_0) * one_minus_cos_squared) * (one_minus_cos_squared * one_minus_cos);

		if (random_float(seed) < F_r) {
			direction = direction_reflected;
		} else {
			direction = direction_refracted;
		}
	}

	ray_buffer_extend.origin[index_out]    = hit_point;
	ray_buffer_extend.direction[index_out] = direction;

	ray_buffer_extend.pixel_index[index_out] = ray_pixel_index;
	ray_buffer_extend.throughput[index_out]  = ray_throughput;

	ray_buffer_extend.last_material_type[index_out] = char(Material::Type::DIELECTRIC);
}

__device__ float beckmann_g1(const float3 & v, const float3 & m, const float3 & n, float alpha) {
	float v_dot_n = dot(v, n);
	if (dot(v, m) / v_dot_n <= 0.0f) return 0.0f;

	float tan_theta_v = sqrt(1.0f - v_dot_n*v_dot_n) / v_dot_n; // tan(acos(x)) = sqrt(1 - x^2) / x
	float a = 1.0f / (alpha * (tan_theta_v));
	
	// Rational approximation
	if (a < 1.6f) {
		return (3.535f * a + 2.181f * a*a) / (1.0f + 2.276f * a + 2.577f * a*a);
	} else {
		return 1.0f;
	}
}

extern "C" __global__ void kernel_shade_glossy(int rand_seed, int bounce) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= N_glossy) return;

	float3 direction_in = -ray_buffer_shade_glossy.direction[index];

	int   ray_triangle_id = ray_buffer_shade_glossy.triangle_id[index];
	float ray_u = ray_buffer_shade_glossy.u[index];
	float ray_v = ray_buffer_shade_glossy.v[index];

	int    ray_pixel_index = ray_buffer_shade_glossy.pixel_index[index];
	float3 ray_throughput  = ray_buffer_shade_glossy.throughput[index];
	
	Material::Type ray_last_material_type = Material::Type(ray_buffer_shade_glossy.last_material_type[index]);
	
	int x = ray_pixel_index % SCREEN_WIDTH;
	int y = ray_pixel_index / SCREEN_WIDTH; 

	ASSERT(ray_triangle_id != -1, "Ray must have hit something for this Kernel to be invoked!");

	unsigned seed = (ray_pixel_index + rand_seed * 312080213) * 781939187;

	const Material & material = materials[triangles_material_id[ray_triangle_id]];

	if (light_count > 0) {
		if (material.is_light()) {
			if (ray_last_material_type == Material::Type::DIELECTRIC) {
				frame_buffer_add(x, y, ray_throughput * material.emittance);
			}

			return;
		}

		// int shadow_ray_index = atomic_agg_inc(&N_shadow);

		// shadow_ray_buffer.triangle_id[shadow_ray_index] = ray_triangle_id;
		// shadow_ray_buffer.u[shadow_ray_index] = ray_u;
		// shadow_ray_buffer.v[shadow_ray_index] = ray_v;

		// shadow_ray_buffer.pixel_index[shadow_ray_index] = ray_pixel_index;
		// shadow_ray_buffer.throughput[shadow_ray_index]  = ray_throughput;
	}

	// Russian Roulette termination
	if (bounce > 3) {
		float one_minus_p = fmaxf(ray_throughput.x, fmaxf(ray_throughput.y, ray_throughput.z));
		if (random_float(seed) > one_minus_p) {
			return;
		}

		ray_throughput /= one_minus_p;
	}

	float3 hit_point     = barycentric(ray_u, ray_v, triangles_position0 [ray_triangle_id], triangles_position_edge1 [ray_triangle_id], triangles_position_edge2 [ray_triangle_id]);
	float3 hit_normal    = barycentric(ray_u, ray_v, triangles_normal0   [ray_triangle_id], triangles_normal_edge1   [ray_triangle_id], triangles_normal_edge2   [ray_triangle_id]);
	float2 hit_tex_coord = barycentric(ray_u, ray_v, triangles_tex_coord0[ray_triangle_id], triangles_tex_coord_edge1[ray_triangle_id], triangles_tex_coord_edge2[ray_triangle_id]);

	if (dot(direction_in, hit_normal) < 0.0f) hit_normal = -hit_normal;

	float alpha = material.roughness; // (1.2f - 0.2f * sqrt(dot(direction_in, hit_normal))) * material.alpha;
	
	// Sample normal distribution in spherical coordinates
	float theta = atan(sqrt(-alpha * alpha * log(1.0f - random_float(seed))));
	float phi   = TWO_PI * random_float(seed);

	float sin_theta, cos_theta;
	float sin_phi,   cos_phi;

	sincos(theta, &sin_theta, &cos_theta);
	sincos(phi,   &sin_phi,   &cos_phi);

	// Convert from spherical coordinates to cartesian coordinates
	float3 micro_normal_local = make_float3(sin_theta * cos_phi, sin_theta * sin_phi, cos_theta);

	float3 hit_tangent, hit_binormal;
	orthonormal_basis(hit_normal, hit_tangent, hit_binormal);

	float3 micro_normal_world = local_to_world(micro_normal_local, hit_tangent, hit_binormal, hit_normal);

	float3 direction_out = reflect(-direction_in, micro_normal_world);

	float g = 
		beckmann_g1(direction_in,  micro_normal_world, hit_normal, alpha) * 
		beckmann_g1(direction_out, micro_normal_world, hit_normal, alpha);
	float weight = abs(dot(direction_in, micro_normal_world)) * g / abs(dot(direction_in, hit_normal) * dot(micro_normal_world, hit_normal));

	int index_out = atomic_agg_inc(&N_ext);

	ray_buffer_extend.origin[index_out]    = hit_point;
	ray_buffer_extend.direction[index_out] = direction_out;

	ray_buffer_extend.pixel_index[index_out] = ray_pixel_index;
	ray_buffer_extend.throughput[index_out]  = ray_throughput * material.albedo(hit_tex_coord.x, hit_tex_coord.y) * weight;

	ray_buffer_extend.last_material_type[index_out] = char(Material::Type::GLOSSY);
}

extern "C" __global__ void kernel_connect(int rand_seed) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= N_shadow) return;

	int   ray_triangle_id = shadow_ray_buffer.triangle_id[index];
	float ray_u = shadow_ray_buffer.u[index];
	float ray_v = shadow_ray_buffer.v[index];

	int    ray_pixel_index = shadow_ray_buffer.pixel_index[index];
	float3 ray_throughput  = shadow_ray_buffer.throughput[index];

	unsigned seed = (ray_pixel_index + rand_seed * 390292093) * 162898261;

	// Pick a random light emitting triangle
	int light_triangle_id = light_indices[rand_xorshift(seed) % light_count];

	ASSERT(length(materials[triangles_material_id[light_triangle_id]].emittance) > 0.0f, "Material was not emissive!\n");

	// Pick a random point on the triangle using random barycentric coordinates
	float u = random_float(seed);
	float v = random_float(seed);

	if (u + v > 1.0f) {
		u = 1.0f - u;
		v = 1.0f - v;
	}

	float3 random_point_on_light = barycentric(u,     v,     triangles_position0[light_triangle_id], triangles_position_edge1[light_triangle_id], triangles_position_edge2[light_triangle_id]);
	float3 hit_point             = barycentric(ray_u, ray_v, triangles_position0[ray_triangle_id],   triangles_position_edge1[ray_triangle_id],   triangles_position_edge2[ray_triangle_id]);
	float3 hit_normal            = barycentric(ray_u, ray_v, triangles_normal0  [ray_triangle_id],   triangles_normal_edge1  [ray_triangle_id],   triangles_normal_edge2  [ray_triangle_id]);

	float3 to_light = random_point_on_light - hit_point;
	float distance_to_light_squared = dot(to_light, to_light);
	float distance_to_light         = sqrtf(distance_to_light_squared);

	// Normalize the vector to the light
	to_light /= distance_to_light;

	float3 light_normal = barycentric(u, v, triangles_normal0[light_triangle_id], triangles_normal_edge1[light_triangle_id], triangles_normal_edge2[light_triangle_id]);

	float cos_o = -dot(to_light, light_normal);
	float cos_i =  dot(to_light, hit_normal);

	if (cos_o > 0.0f && cos_i > 0.0f) {
		Ray shadow_ray;
		shadow_ray.origin    = hit_point;
		shadow_ray.direction = to_light;
		shadow_ray.direction_inv = make_float3(
			1.0f / shadow_ray.direction.x, 
			1.0f / shadow_ray.direction.y, 
			1.0f / shadow_ray.direction.z
		);

		// Check if the light is obstructed by any other object in the scene
		if (!bvh_intersect(shadow_ray, distance_to_light - EPSILON)) {
			const Material & hit_material   = materials[triangles_material_id[ray_triangle_id]];
			const Material & light_material = materials[triangles_material_id[light_triangle_id]];

			float2 hit_tex_coord = barycentric(ray_u, ray_v, triangles_tex_coord0[ray_triangle_id], triangles_tex_coord_edge1[ray_triangle_id], triangles_tex_coord_edge2[ray_triangle_id]);

			float3 brdf = hit_material.albedo(hit_tex_coord.x, hit_tex_coord.y) * ONE_OVER_PI;

			float light_area = 0.5f * length(cross(
				triangles_position_edge1[light_triangle_id], 
				triangles_position_edge2[light_triangle_id]
			));
			float solid_angle = (cos_o * light_area) / distance_to_light_squared;

			int x = ray_pixel_index % SCREEN_WIDTH;
			int y = ray_pixel_index / SCREEN_WIDTH; 
		
			frame_buffer_add(x, y, ray_throughput * brdf * light_count * light_material.emittance * solid_angle * cos_i);
		}
	}
}

extern "C" __global__ void kernel_accumulate(float frames_since_camera_moved) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	float4 colour;
	surf2Dread<float4>(&colour, frame_buffer, x * sizeof(float4), y);
	
	float4 colour_out;
	if (frames_since_camera_moved > 0.0f) {
		float4 prev;
		surf2Dread<float4>(&prev, accumulator, x * sizeof(float4), y);

		// Take average over n samples by weighing the current content of the framebuffer by (n-1) and the new sample by 1
		colour_out = (prev * (frames_since_camera_moved - 1.0f) + colour) / frames_since_camera_moved;
	} else {
		colour_out = colour;
	}

	surf2Dwrite<float4>(colour_out, accumulator, x * sizeof(float4), y, hipBoundaryModeClamp);

	// Clear frame buffer for next frame
	surf2Dwrite<float4>(make_float4(0.0f, 0.0f, 0.0f, 1.0f), frame_buffer, x * sizeof(float4), y, hipBoundaryModeClamp);
}
