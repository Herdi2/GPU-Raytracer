#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"
#include "cuda_math.h"

#include <corecrt_math.h>

#include "../Common.h"

//#define ASSERT(proposition, fmt, ...) { if (!(proposition)) printf(fmt, __VA_ARGS__); assert(proposition); }
#define ASSERT(proposition, fmt, ...) { }

surface<void, 2> frame_buffer;
surface<void, 2> accumulator;

#define USE_IMPORTANCE_SAMPLING true

// Based on: http://www.reedbeta.com/blog/quick-and-easy-gpu-random-numbers-in-d3d11/
__device__ unsigned rand_xorshift(unsigned & seed) {
    seed ^= (seed << 13);
    seed ^= (seed >> 17);
	seed ^= (seed << 5);
	
    return seed;
}

__device__ float random_float(unsigned & seed) {
	const float one_over_max_unsigned = 2.3283064365387e-10f;
	return float(rand_xorshift(seed)) * one_over_max_unsigned;
}

template<typename T>
__device__ T barycentric(float u, float v, const T & base, const T & edge_1, const T & edge_2) {
	return base + u * edge_1 + v * edge_2;
}

struct Material;

__device__ Material            * materials;
__device__ hipTextureObject_t * textures;

struct Material {
	enum Type : char{
		DIFFUSE    = 0,
		DIELECTRIC = 1,
		GLOSSY     = 2
	};

	Type type;

	float3 diffuse;
	int texture_id;

	float3 emittance;

	float index_of_refraction;

	float roughness;

	__device__ float3 albedo(float u, float v) const {
		if (texture_id == -1) return diffuse;

		float4 tex_colour;

		for (int i = 0; i < MAX_TEXTURES; i++) {
			if (texture_id == i) {
				tex_colour = tex2D<float4>(textures[i], u, v);
			}
		}

		return diffuse * make_float3(tex_colour);
	}

	__device__ bool is_light() const {
		return dot(emittance, emittance) > 0.0f;
	}
};

__device__ int     light_count;
__device__ int   * light_indices;
__device__ float * light_areas;
__device__ float total_light_area;

struct Ray {
	float3 origin;
	float3 direction;
	float3 direction_inv;
};

struct RayHit {
	float t = INFINITY;
	float u, v;

	int triangle_id;
};

struct AABB {
	float3 min;
	float3 max;

	__device__ inline bool intersects(const Ray & ray, float max_distance) const {
		float3 t0 = (min - ray.origin) * ray.direction_inv;
		float3 t1 = (max - ray.origin) * ray.direction_inv;
		
		float3 t_min = fminf(t0, t1);
		float3 t_max = fmaxf(t0, t1);
		
		float t_near = fmaxf(fmaxf(EPSILON,      t_min.x), fmaxf(t_min.y, t_min.z));
		float t_far  = fminf(fminf(max_distance, t_max.x), fminf(t_max.y, t_max.z));
	
		return t_near < t_far;
	}
};

__device__ float3 * triangles_position0;
__device__ float3 * triangles_position_edge1;
__device__ float3 * triangles_position_edge2;

__device__ float3 * triangles_normal0;
__device__ float3 * triangles_normal_edge1;
__device__ float3 * triangles_normal_edge2; 
 
__device__ float2 * triangles_tex_coord0;
__device__ float2 * triangles_tex_coord_edge1;
__device__ float2 * triangles_tex_coord_edge2;

__device__ int * triangles_material_id;

struct BVHNode {
	AABB aabb;
	union {
		int left;
		int first;
	};
	int count;

	__device__ inline bool is_leaf() const {
		return (count & (~BVH_AXIS_MASK)) > 0;
	}

	__device__ inline bool should_visit_left_first(const Ray & ray) const {
#if BVH_TRAVERSAL_STRATEGY == BVH_TRAVERSE_TREE_NAIVE
		return true; // Naive always goes left first
#elif BVH_TRAVERSAL_STRATEGY == BVH_TRAVERSE_TREE_ORDERED
		switch (count & BVH_AXIS_MASK) {
			case BVH_AXIS_X_BITS: return ray.direction.x > 0.0f;
			case BVH_AXIS_Y_BITS: return ray.direction.y > 0.0f;
			case BVH_AXIS_Z_BITS: return ray.direction.z > 0.0f;
		}
#endif
	}
};

__device__ BVHNode * bvh_nodes;

__device__ void triangle_trace(int triangle_id, const Ray & ray, RayHit & ray_hit) {
	const float3 & position0      = triangles_position0[triangle_id];
	const float3 & position_edge1 = triangles_position_edge1[triangle_id];
	const float3 & position_edge2 = triangles_position_edge2[triangle_id];

	float3 h = cross(ray.direction, position_edge2);
	float  a = dot(position_edge1, h);

	float  f = 1.0f / a;
	float3 s = ray.origin - position0;
	float  u = f * dot(s, h);

	if (u < 0.0f || u > 1.0f) return;

	float3 q = cross(s, position_edge1);
	float  v = f * dot(ray.direction, q);

	if (v < 0.0f || u + v > 1.0f) return;

	float t = f * dot(position_edge2, q);

	if (t < EPSILON || t >= ray_hit.t) return;

	ray_hit.t = t;
	ray_hit.u = u;
	ray_hit.v = v;
	ray_hit.triangle_id = triangle_id;
}

__device__ bool triangle_intersect(int triangle_id, const Ray & ray, float max_distance) {
	const float3 & position0      = triangles_position0[triangle_id];
	const float3 & position_edge1 = triangles_position_edge1[triangle_id];
	const float3 & position_edge2 = triangles_position_edge2[triangle_id];

	float3 h = cross(ray.direction, position_edge2);
	float  a = dot(position_edge1, h);

	float  f = 1.0f / a;
	float3 s = ray.origin - position0;
	float  u = f * dot(s, h);

	if (u < 0.0f || u > 1.0f) return false;

	float3 q = cross(s, position_edge1);
	float  v = f * dot(ray.direction, q);

	if (v < 0.0f || u + v > 1.0f) return false;

	float t = f * dot(position_edge2, q);

	if (t < EPSILON || t >= max_distance) return false;

	return true;
}

__device__ void bvh_trace(const Ray & ray, RayHit & ray_hit) {
	int stack[64];
	int stack_size = 1;

	// Push root on stack
	stack[0] = 0;

	while (stack_size > 0) {
		// Pop Node of the stack
		const BVHNode & node = bvh_nodes[stack[--stack_size]];

		if (node.aabb.intersects(ray, ray_hit.t)) {
			if (node.is_leaf()) {
				for (int i = node.first; i < node.first + node.count; i++) {
					triangle_trace(i, ray, ray_hit);
				}
			} else {
				if (node.should_visit_left_first(ray)) {
					stack[stack_size++] = node.left + 1;
					stack[stack_size++] = node.left;
				} else {
					stack[stack_size++] = node.left;
					stack[stack_size++] = node.left + 1;
				}
			}
		}
	}
}

__device__ bool bvh_intersect(const Ray & ray, float max_distance) {
	int stack[64];
	int stack_size = 1;

	// Push root on stack
	stack[0] = 0;

	while (stack_size > 0) {
		// Pop Node of the stack
		const BVHNode & node = bvh_nodes[stack[--stack_size]];

		if (node.aabb.intersects(ray, max_distance)) {
			if (node.is_leaf()) {
				for (int i = node.first; i < node.first + node.count; i++) {
					if (triangle_intersect(i, ray, max_distance)) {
						return true;
					}
				}
			} else {
				if (node.should_visit_left_first(ray)) {
					stack[stack_size++] = node.left + 1;
					stack[stack_size++] = node.left;
				} else {
					stack[stack_size++] = node.left;
					stack[stack_size++] = node.left + 1;
				}
			}
		}
	}

	return false;
}

__device__ int      sky_size;
__device__ float3 * sky_data;

__device__ float3 sample_sky(const float3 & direction) {
	// Formulas as described on https://www.pauldebevec.com/Probes/
    float r = 0.5f * ONE_OVER_PI * acos(direction.z) * rsqrt(direction.x*direction.x + direction.y*direction.y);

	float u = direction.x * r + 0.5f;
	float v = direction.y * r + 0.5f;

	// Convert to pixel coordinates
	int x = int(u * sky_size);
	int y = int(v * sky_size);

	int index = x + y * sky_size;
	index = max(index, 0);
	index = min(index, sky_size * sky_size);

	return sky_data[index];
}

__device__ void orthonormal_basis(const float3 & normal, float3 & tangent, float3 & binormal) {
	// Calculate a tangent vector from the normal vector
	if (fabsf(normal.x) > 0.99f) {
		tangent = make_float3(-normal.z, 0.0f, normal.x) * rsqrt(normal.x * normal.x + normal.z * normal.z);
	} else {
		tangent = make_float3(0.0f, normal.z, -normal.y) * rsqrt(normal.y * normal.y + normal.z * normal.z);
	}

	// The binormal is perpendicular to both the normal and tangent vectors
	binormal = cross(normal, tangent);
}

__device__ float3 local_to_world(const float3 & vector, const float3 & tangent, const float3 & binormal, const float3 & normal) {
	return make_float3(
		tangent.x * vector.x + binormal.x * vector.y + normal.x * vector.z, 
		tangent.y * vector.x + binormal.y * vector.y + normal.y * vector.z, 
		tangent.z * vector.x + binormal.z * vector.y + normal.z * vector.z
	);
}

__device__ float3 world_to_local(const float3 & vector, const float3 & tangent, const float3 & binormal, const float3 & normal) {
	return make_float3(dot(tangent, vector), dot(binormal, vector), dot(normal, vector));
}

__device__ float3 cosine_weighted_diffuse_reflection(unsigned & seed, const float3 & normal) {
	float r0 = random_float(seed);
	float r1 = random_float(seed);

	float sin_theta, cos_theta;
	sincos(TWO_PI * r1, &sin_theta, &cos_theta);

	float r = sqrtf(r0);
	float x = r * cos_theta;
	float y = r * sin_theta;
	
	float3 direction = normalize(make_float3(x, y, sqrtf(1.0f - r0)));
	
	float3 tangent, binormal;
	orthonormal_basis(normal, tangent, binormal);

	// Multiply the direction with the TBN matrix
	direction = local_to_world(direction, tangent, binormal, normal);

	ASSERT(dot(direction, normal) > -1e-5, "Invalid dot: dot = %f, direction = (%f, %f, %f), normal = (%f, %f, %f)\n", 
		dot(direction, normal), direction.x, direction.y, direction.z, normal.x, normal.y, normal.z
	);

	return direction;
}

// Based on: https://devblogs.nvidia.com/cuda-pro-tip-optimized-filtering-warp-aggregated-atomics/
__device__ inline int atomic_agg_inc(int * ctr) {
	int mask   = __ballot(1);
	int leader = __ffs(mask) - 1;
	int laneid = threadIdx.x % 32;
	
	int res;
	if (laneid == leader) {
		res = atomicAdd(ctr, __popc(mask));
	}

	res = __shfl(res, leader);
	return res + __popc(mask & ((1 << laneid) - 1));
}

__device__ void frame_buffer_add(int x, int y, const float3 & colour) {
	float4 prev;
	surf2Dread<float4>(&prev, frame_buffer, x * sizeof(float4), y);
	
	surf2Dwrite<float4>(prev + make_float4(colour, 0.0f), frame_buffer, x * sizeof(float4), y, hipBoundaryModeClamp);
}

struct RayBuffer {
	float3 * origin;
	float3 * direction;
	
	int   * triangle_id;
	float * u;
	float * v;

	int    * pixel_index;
	float3 * throughput;

	char * last_material_type;
};

__device__ RayBuffer ray_buffer_extend;
__device__ RayBuffer ray_buffer_shade_diffuse;
__device__ RayBuffer ray_buffer_shade_dielectric;
__device__ RayBuffer ray_buffer_shade_glossy;

struct ShadowRayBuffer {
	int   * triangle_id;
	float * u;
	float * v;

	int    * pixel_index;
	float3 * throughput;
};

__device__ ShadowRayBuffer shadow_ray_buffer;

__device__ int N_ext;
__device__ int N_diffuse;
__device__ int N_dielectric;
__device__ int N_glossy;
__device__ int N_shadow;

extern "C" __global__ void kernel_generate(
	int rand_seed,
	float3 camera_position,
	float3 camera_top_left_corner,
	float3 camera_x_axis,
	float3 camera_y_axis
) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= SCREEN_WIDTH * SCREEN_HEIGHT) return;

	unsigned seed = (index + rand_seed * 199494991) * 949525949;
	
	const int BLOCK_WIDTH  = 16;
	const int BLOCK_HEIGHT = 8;
	const int BLOCK_SIZE   = BLOCK_WIDTH * BLOCK_HEIGHT;

	int block_index = index / BLOCK_SIZE;
	int i = (block_index % (SCREEN_WIDTH / BLOCK_WIDTH)) * BLOCK_WIDTH;
	int j = (block_index / (SCREEN_WIDTH / BLOCK_WIDTH)) * BLOCK_HEIGHT;

	ASSERT(i < SCREEN_WIDTH, "");
	ASSERT(j < SCREEN_HEIGHT, "");

	int k = (index % BLOCK_SIZE) % BLOCK_WIDTH;
	int l = (index % BLOCK_SIZE) / BLOCK_WIDTH;

	ASSERT(k < BLOCK_WIDTH, "");
	ASSERT(l < BLOCK_HEIGHT, "");

	int x = i + k;
	int y = j + l;

	ASSERT(x < SCREEN_WIDTH, "");
	ASSERT(y < SCREEN_HEIGHT, "");

	int pixel_index = x + y * SCREEN_WIDTH;

	// Add random value between 0 and 1 so that after averaging we get anti-aliasing
	float u = x + random_float(seed);
	float v = y + random_float(seed);

	ASSERT(pixel_index < SCREEN_WIDTH * SCREEN_HEIGHT, "Pixel should be on screen");

	// Create primary Ray that starts at the Camera's position and goes trough the current pixel
	ray_buffer_extend.origin[index]    = camera_position;
	ray_buffer_extend.direction[index] = normalize(camera_top_left_corner
		+ u * camera_x_axis
		+ v * camera_y_axis
	);
	
	ray_buffer_extend.pixel_index[index] = pixel_index;
	ray_buffer_extend.throughput[index]  = make_float3(1.0f);

	ray_buffer_extend.last_material_type[index] = char(Material::Type::DIELECTRIC);
}

extern "C" __global__ void kernel_extend() {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= N_ext) return;

	float3 ray_origin    = ray_buffer_extend.origin[index];
	float3 ray_direction = ray_buffer_extend.direction[index];
	
	Ray ray;
	ray.origin    = ray_origin;
	ray.direction = ray_direction;
	ray.direction_inv = make_float3(
		1.0f / ray.direction.x, 
		1.0f / ray.direction.y, 
		1.0f / ray.direction.z
	);

	RayHit hit;
	bvh_trace(ray, hit);

	if (hit.t == INFINITY) {
		int ray_pixel_index = ray_buffer_extend.pixel_index[index];

		int x = ray_pixel_index % SCREEN_WIDTH;
		int y = ray_pixel_index / SCREEN_WIDTH; 

		frame_buffer_add(x, y, ray_buffer_extend.throughput[index] * sample_sky(ray_direction));

		return;
	}

	Material::Type material_type = materials[triangles_material_id[hit.triangle_id]].type;
	
	if (material_type == Material::Type::DIFFUSE) {
		int index_out = atomic_agg_inc(&N_diffuse);

		// ray_buffer_shade_diffuse.direction[index_out] = ray_direction;

		ray_buffer_shade_diffuse.triangle_id[index_out] = hit.triangle_id;
		ray_buffer_shade_diffuse.u[index_out] = hit.u;
		ray_buffer_shade_diffuse.v[index_out] = hit.v;

		ray_buffer_shade_diffuse.pixel_index[index_out] = ray_buffer_extend.pixel_index[index];
		ray_buffer_shade_diffuse.throughput[index_out]  = ray_buffer_extend.throughput[index];

		ray_buffer_shade_diffuse.last_material_type[index_out] = ray_buffer_extend.last_material_type[index];
	} else if (material_type == Material::Type::DIELECTRIC) {
		int index_out = atomic_agg_inc(&N_dielectric);

		ray_buffer_shade_dielectric.direction[index_out] = ray_direction;

		ray_buffer_shade_dielectric.triangle_id[index_out] = hit.triangle_id;
		ray_buffer_shade_dielectric.u[index_out] = hit.u;
		ray_buffer_shade_dielectric.v[index_out] = hit.v;

		ray_buffer_shade_dielectric.pixel_index[index_out] = ray_buffer_extend.pixel_index[index];
		ray_buffer_shade_dielectric.throughput[index_out]  = ray_buffer_extend.throughput[index];

		ray_buffer_shade_dielectric.last_material_type[index_out] = ray_buffer_extend.last_material_type[index];
	} else if (material_type == Material::Type::GLOSSY) {
		int index_out = atomic_agg_inc(&N_glossy);

		ray_buffer_shade_glossy.direction[index_out] = ray_direction;

		ray_buffer_shade_glossy.triangle_id[index_out] = hit.triangle_id;
		ray_buffer_shade_glossy.u[index_out] = hit.u;
		ray_buffer_shade_glossy.v[index_out] = hit.v;

		ray_buffer_shade_glossy.pixel_index[index_out] = ray_buffer_extend.pixel_index[index];
		ray_buffer_shade_glossy.throughput[index_out]  = ray_buffer_extend.throughput[index];

		ray_buffer_shade_glossy.last_material_type[index_out] = ray_buffer_extend.last_material_type[index];
	}
}

extern "C" __global__ void kernel_shade_diffuse(int rand_seed, int bounce) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= N_diffuse) return;

	// float3 ray_direction = ray_buffer_shade_diffuse.direction[index];

	int   ray_triangle_id = ray_buffer_shade_diffuse.triangle_id[index];
	float ray_u = ray_buffer_shade_diffuse.u[index];
	float ray_v = ray_buffer_shade_diffuse.v[index];

	int    ray_pixel_index = ray_buffer_shade_diffuse.pixel_index[index];
	float3 ray_throughput  = ray_buffer_shade_diffuse.throughput[index];

	Material::Type ray_last_material_type = Material::Type(ray_buffer_shade_diffuse.last_material_type[index]);

	int x = ray_pixel_index % SCREEN_WIDTH;
	int y = ray_pixel_index / SCREEN_WIDTH; 

	ASSERT(ray_triangle_id != -1, "Ray must have hit something for this Kernel to be invoked!");

	unsigned seed = (ray_pixel_index + rand_seed * 312080213) * 781939187;

	const Material & material = materials[triangles_material_id[ray_triangle_id]];

	if (light_count > 0) {
		if (material.is_light()) {
			if (ray_last_material_type == Material::Type::DIELECTRIC) {
				frame_buffer_add(x, y, ray_throughput * material.emittance);
			}

			return;
		}

		int shadow_ray_index = atomic_agg_inc(&N_shadow);

		shadow_ray_buffer.triangle_id[shadow_ray_index] = ray_triangle_id;
		shadow_ray_buffer.u[shadow_ray_index] = ray_u;
		shadow_ray_buffer.v[shadow_ray_index] = ray_v;

		shadow_ray_buffer.pixel_index[shadow_ray_index] = ray_pixel_index;
		shadow_ray_buffer.throughput[shadow_ray_index]  = ray_throughput;
	}

	// Russian Roulette termination
	if (bounce > 3) {
		float one_minus_p = fmaxf(ray_throughput.x, fmaxf(ray_throughput.y, ray_throughput.z));
		if (random_float(seed) > one_minus_p) {
			return;
		}

		ray_throughput /= one_minus_p;
	}

	float3 hit_point     = barycentric(ray_u, ray_v, triangles_position0 [ray_triangle_id], triangles_position_edge1 [ray_triangle_id], triangles_position_edge2 [ray_triangle_id]);
	float3 hit_normal    = barycentric(ray_u, ray_v, triangles_normal0   [ray_triangle_id], triangles_normal_edge1   [ray_triangle_id], triangles_normal_edge2   [ray_triangle_id]);
	float2 hit_tex_coord = barycentric(ray_u, ray_v, triangles_tex_coord0[ray_triangle_id], triangles_tex_coord_edge1[ray_triangle_id], triangles_tex_coord_edge2[ray_triangle_id]);

	// if (dot(ray_direction, hit_normal) > 0.0f) hit_normal = -hit_normal;

	int index_out = atomic_agg_inc(&N_ext);

	ray_buffer_extend.origin[index_out]    = hit_point;
	ray_buffer_extend.direction[index_out] = cosine_weighted_diffuse_reflection(seed, hit_normal);

	ray_buffer_extend.pixel_index[index_out] = ray_pixel_index;
	ray_buffer_extend.throughput[index_out]  = ray_throughput * material.albedo(hit_tex_coord.x, hit_tex_coord.y);

	ray_buffer_extend.last_material_type[index_out] = char(Material::Type::DIFFUSE);
}

extern "C" __global__ void kernel_shade_dielectric(int rand_seed, int bounce) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= N_dielectric) return;

	float3 ray_direction = ray_buffer_shade_dielectric.direction[index];

	int   ray_triangle_id = ray_buffer_shade_dielectric.triangle_id[index];
	float ray_u = ray_buffer_shade_dielectric.u[index];
	float ray_v = ray_buffer_shade_dielectric.v[index];

	int    ray_pixel_index = ray_buffer_shade_dielectric.pixel_index[index];
	float3 ray_throughput  = ray_buffer_shade_dielectric.throughput[index];

	Material::Type ray_last_material_type = Material::Type(ray_buffer_shade_dielectric.last_material_type[index]);

	int x = ray_pixel_index % SCREEN_WIDTH;
	int y = ray_pixel_index / SCREEN_WIDTH; 

	ASSERT(ray_triangle_id != -1, "Ray must have hit something for this Kernel to be invoked!");

	unsigned seed = (ray_pixel_index + rand_seed * 312080213) * 781939187;

	const Material & material = materials[triangles_material_id[ray_triangle_id]];

	if (light_count > 0) {
		if (material.is_light()) {
			if (ray_last_material_type == Material::Type::DIELECTRIC) {
				frame_buffer_add(x, y, ray_throughput * material.emittance);
			}

			return;
		}
	}

	// Russian Roulette termination
	if (bounce > 3) {
		float one_minus_p = fmaxf(ray_throughput.x, fmaxf(ray_throughput.y, ray_throughput.z));
		if (random_float(seed) > one_minus_p) {
			return;
		}

		ray_throughput /= one_minus_p;
	}

	float3 hit_point     = barycentric(ray_u, ray_v, triangles_position0 [ray_triangle_id], triangles_position_edge1 [ray_triangle_id], triangles_position_edge2 [ray_triangle_id]);
	float3 hit_normal    = barycentric(ray_u, ray_v, triangles_normal0   [ray_triangle_id], triangles_normal_edge1   [ray_triangle_id], triangles_normal_edge2   [ray_triangle_id]);
	float2 hit_tex_coord = barycentric(ray_u, ray_v, triangles_tex_coord0[ray_triangle_id], triangles_tex_coord_edge1[ray_triangle_id], triangles_tex_coord_edge2[ray_triangle_id]);

	// if (dot(ray_direction, hit_normal) > 0.0f) hit_normal = -hit_normal;

	int index_out = atomic_agg_inc(&N_ext);

	float3 direction;
	float3 direction_reflected = reflect(ray_direction, hit_normal);

	float3 normal;
	float  cos_theta;

	float n_1;
	float n_2;

	float dir_dot_normal = dot(ray_direction, hit_normal);
	if (dir_dot_normal < 0.0f) { 
		// Entering material		
		n_1 = 1.0f;
		n_2 = material.index_of_refraction;

		normal    =  hit_normal;
		cos_theta = -dir_dot_normal;
	} else { 
		// Leaving material
		n_1 = material.index_of_refraction;
		n_2 = 1.0f;

		normal    = -hit_normal;
		cos_theta =  dir_dot_normal;
	}

	float eta = n_1 / n_2;
	float k = 1.0f - eta*eta * (1.0f - cos_theta*cos_theta);

	if (k < 0.0f) {
		direction = direction_reflected;
	} else {
		float3 direction_refracted = normalize(eta * ray_direction + (eta * cos_theta - sqrtf(k)) * hit_normal);

		// Use Schlick's Approximation
		float r_0 = (n_1 - n_2) / (n_1 + n_2);
		r_0 *= r_0;

		if (n_1 > n_2) {
			cos_theta = -dot(direction_refracted, normal);
		}

		float one_minus_cos         = 1.0f - cos_theta;
		float one_minus_cos_squared = one_minus_cos * one_minus_cos;

		float F_r = r_0 + ((1.0f - r_0) * one_minus_cos_squared) * (one_minus_cos_squared * one_minus_cos);

		if (random_float(seed) < F_r) {
			direction = direction_reflected;
		} else {
			direction = direction_refracted;
		}
	}

	ray_buffer_extend.origin[index_out]    = hit_point;
	ray_buffer_extend.direction[index_out] = direction;

	ray_buffer_extend.pixel_index[index_out] = ray_pixel_index;
	ray_buffer_extend.throughput[index_out]  = ray_throughput;

	ray_buffer_extend.last_material_type[index_out] = char(Material::Type::DIELECTRIC);
}

__device__ float beckmann_g1(const float3 & v, const float3 & m, const float3 & n, float alpha) {
	float v_dot_n = dot(v, n);
	if (dot(v, m) / v_dot_n <= 0.0f) return 0.0f;

	float tan_theta_v = sqrt(1.0f - v_dot_n*v_dot_n) / v_dot_n; // tan(acos(x)) = sqrt(1 - x^2) / x
	float a = 1.0f / (alpha * (tan_theta_v));
	
	// Rational approximation
	if (a < 1.6f) {
		return (3.535f * a + 2.181f * a*a) / (1.0f + 2.276f * a + 2.577f * a*a);
	} else {
		return 1.0f;
	}
}

extern "C" __global__ void kernel_shade_glossy(int rand_seed, int bounce) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= N_glossy) return;

	float3 direction_in = -ray_buffer_shade_glossy.direction[index];

	int   ray_triangle_id = ray_buffer_shade_glossy.triangle_id[index];
	float ray_u = ray_buffer_shade_glossy.u[index];
	float ray_v = ray_buffer_shade_glossy.v[index];

	int    ray_pixel_index = ray_buffer_shade_glossy.pixel_index[index];
	float3 ray_throughput  = ray_buffer_shade_glossy.throughput[index];
	
	Material::Type ray_last_material_type = Material::Type(ray_buffer_shade_glossy.last_material_type[index]);
	
	int x = ray_pixel_index % SCREEN_WIDTH;
	int y = ray_pixel_index / SCREEN_WIDTH; 

	ASSERT(ray_triangle_id != -1, "Ray must have hit something for this Kernel to be invoked!");

	unsigned seed = (ray_pixel_index + rand_seed * 312080213) * 781939187;

	const Material & material = materials[triangles_material_id[ray_triangle_id]];

	if (light_count > 0) {
		if (material.is_light()) {
			if (ray_last_material_type == Material::Type::DIELECTRIC) {
				frame_buffer_add(x, y, ray_throughput * material.emittance);
			}

			return;
		}

		// int shadow_ray_index = atomic_agg_inc(&N_shadow);

		// shadow_ray_buffer.triangle_id[shadow_ray_index] = ray_triangle_id;
		// shadow_ray_buffer.u[shadow_ray_index] = ray_u;
		// shadow_ray_buffer.v[shadow_ray_index] = ray_v;

		// shadow_ray_buffer.pixel_index[shadow_ray_index] = ray_pixel_index;
		// shadow_ray_buffer.throughput[shadow_ray_index]  = ray_throughput;
	}

	// Russian Roulette termination
	if (bounce > 3) {
		float one_minus_p = fmaxf(ray_throughput.x, fmaxf(ray_throughput.y, ray_throughput.z));
		if (random_float(seed) > one_minus_p) {
			return;
		}

		ray_throughput /= one_minus_p;
	}

	float3 hit_point     = barycentric(ray_u, ray_v, triangles_position0 [ray_triangle_id], triangles_position_edge1 [ray_triangle_id], triangles_position_edge2 [ray_triangle_id]);
	float3 hit_normal    = barycentric(ray_u, ray_v, triangles_normal0   [ray_triangle_id], triangles_normal_edge1   [ray_triangle_id], triangles_normal_edge2   [ray_triangle_id]);
	float2 hit_tex_coord = barycentric(ray_u, ray_v, triangles_tex_coord0[ray_triangle_id], triangles_tex_coord_edge1[ray_triangle_id], triangles_tex_coord_edge2[ray_triangle_id]);

	if (dot(direction_in, hit_normal) < 0.0f) hit_normal = -hit_normal;

	float alpha = material.roughness; // (1.2f - 0.2f * sqrt(dot(direction_in, hit_normal))) * material.alpha;
	
	// Sample normal distribution in spherical coordinates
	float theta = atan(sqrt(-alpha * alpha * log(1.0f - random_float(seed))));
	float phi   = TWO_PI * random_float(seed);

	float sin_theta, cos_theta;
	float sin_phi,   cos_phi;

	sincos(theta, &sin_theta, &cos_theta);
	sincos(phi,   &sin_phi,   &cos_phi);

	// Convert from spherical coordinates to cartesian coordinates
	float3 micro_normal_local = make_float3(sin_theta * cos_phi, sin_theta * sin_phi, cos_theta);

	float3 hit_tangent, hit_binormal;
	orthonormal_basis(hit_normal, hit_tangent, hit_binormal);

	float3 micro_normal_world = local_to_world(micro_normal_local, hit_tangent, hit_binormal, hit_normal);

	float3 direction_out = reflect(-direction_in, micro_normal_world);

	float g = 
		beckmann_g1(direction_in,  micro_normal_world, hit_normal, alpha) * 
		beckmann_g1(direction_out, micro_normal_world, hit_normal, alpha);
	float weight = abs(dot(direction_in, micro_normal_world)) * g / abs(dot(direction_in, hit_normal) * dot(micro_normal_world, hit_normal));

	int index_out = atomic_agg_inc(&N_ext);

	ray_buffer_extend.origin[index_out]    = hit_point;
	ray_buffer_extend.direction[index_out] = direction_out;

	ray_buffer_extend.pixel_index[index_out] = ray_pixel_index;
	ray_buffer_extend.throughput[index_out]  = ray_throughput * material.albedo(hit_tex_coord.x, hit_tex_coord.y) * weight;

	ray_buffer_extend.last_material_type[index_out] = char(Material::Type::GLOSSY);
}

extern "C" __global__ void kernel_connect(int rand_seed) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= N_shadow) return;

	int   ray_triangle_id = shadow_ray_buffer.triangle_id[index];
	float ray_u = shadow_ray_buffer.u[index];
	float ray_v = shadow_ray_buffer.v[index];

	int    ray_pixel_index = shadow_ray_buffer.pixel_index[index];
	float3 ray_throughput  = shadow_ray_buffer.throughput[index];

	unsigned seed = (ray_pixel_index + rand_seed * 390292093) * 162898261;

	// Pick a random light emitting triangle
	int light_triangle_id = light_indices[rand_xorshift(seed) % light_count];

	ASSERT(length(materials[triangles_material_id[light_triangle_id]].emittance) > 0.0f, "Material was not emissive!\n");

	// Pick a random point on the triangle using random barycentric coordinates
	float u = random_float(seed);
	float v = random_float(seed);

	if (u + v > 1.0f) {
		u = 1.0f - u;
		v = 1.0f - v;
	}

	float3 random_point_on_light = barycentric(u,     v,     triangles_position0[light_triangle_id], triangles_position_edge1[light_triangle_id], triangles_position_edge2[light_triangle_id]);
	float3 hit_point             = barycentric(ray_u, ray_v, triangles_position0[ray_triangle_id],   triangles_position_edge1[ray_triangle_id],   triangles_position_edge2[ray_triangle_id]);
	float3 hit_normal            = barycentric(ray_u, ray_v, triangles_normal0  [ray_triangle_id],   triangles_normal_edge1  [ray_triangle_id],   triangles_normal_edge2  [ray_triangle_id]);

	float3 to_light = random_point_on_light - hit_point;
	float distance_to_light_squared = dot(to_light, to_light);
	float distance_to_light         = sqrtf(distance_to_light_squared);

	// Normalize the vector to the light
	to_light /= distance_to_light;

	float3 light_normal = barycentric(u, v, triangles_normal0[light_triangle_id], triangles_normal_edge1[light_triangle_id], triangles_normal_edge2[light_triangle_id]);

	float cos_o = -dot(to_light, light_normal);
	float cos_i =  dot(to_light, hit_normal);

	if (cos_o > 0.0f && cos_i > 0.0f) {
		Ray shadow_ray;
		shadow_ray.origin    = hit_point;
		shadow_ray.direction = to_light;
		shadow_ray.direction_inv = make_float3(
			1.0f / shadow_ray.direction.x, 
			1.0f / shadow_ray.direction.y, 
			1.0f / shadow_ray.direction.z
		);

		// Check if the light is obstructed by any other object in the scene
		if (!bvh_intersect(shadow_ray, distance_to_light - EPSILON)) {
			const Material & hit_material   = materials[triangles_material_id[ray_triangle_id]];
			const Material & light_material = materials[triangles_material_id[light_triangle_id]];

			float2 hit_tex_coord = barycentric(ray_u, ray_v, triangles_tex_coord0[ray_triangle_id], triangles_tex_coord_edge1[ray_triangle_id], triangles_tex_coord_edge2[ray_triangle_id]);

			float3 brdf = hit_material.albedo(hit_tex_coord.x, hit_tex_coord.y) * ONE_OVER_PI;

			float light_area = 0.5f * length(cross(
				triangles_position_edge1[light_triangle_id], 
				triangles_position_edge2[light_triangle_id]
			));
			float solid_angle = (cos_o * light_area) / distance_to_light_squared;

			int x = ray_pixel_index % SCREEN_WIDTH;
			int y = ray_pixel_index / SCREEN_WIDTH; 
		
			frame_buffer_add(x, y, ray_throughput * brdf * light_count * light_material.emittance * solid_angle * cos_i);
		}
	}
}

extern "C" __global__ void kernel_accumulate(float frames_since_camera_moved) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	float4 colour;
	surf2Dread<float4>(&colour, frame_buffer, x * sizeof(float4), y);
	
	float4 colour_out;
	if (frames_since_camera_moved > 0.0f) {
		float4 prev;
		surf2Dread<float4>(&prev, accumulator, x * sizeof(float4), y);

		// Take average over n samples by weighing the current content of the framebuffer by (n-1) and the new sample by 1
		colour_out = (prev * (frames_since_camera_moved - 1.0f) + colour) / frames_since_camera_moved;
	} else {
		colour_out = colour;
	}

	surf2Dwrite<float4>(colour_out, accumulator, x * sizeof(float4), y, hipBoundaryModeClamp);

	// Clear frame buffer for next frame
	surf2Dwrite<float4>(make_float4(0.0f, 0.0f, 0.0f, 1.0f), frame_buffer, x * sizeof(float4), y, hipBoundaryModeClamp);
}
